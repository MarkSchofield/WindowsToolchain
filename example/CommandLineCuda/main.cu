//---------------------------------------------------------------------------------------------------------------------
//
//---------------------------------------------------------------------------------------------------------------------
#include <hip/hip_runtime.h>

#include <exception>
#include <iostream>

int main(int, char**)
{
    try
    {
        int deviceCount = 0;
        hipError_t error_id = hipGetDeviceCount(&deviceCount);
        if (error_id != hipSuccess)
        {
            std::cerr << "hipGetDeviceCount returned " << static_cast<int>(error_id) << ".\n"
                      << " -> " << hipGetErrorString(error_id) << "\n";
            exit(EXIT_FAILURE);
        }

        std::cout << "Detected " << deviceCount << " CUDA capable device(s)\n";
    }
    catch (const std::exception& ex)
    {
        std::cerr << "Exception: " << ex.what();
    }
}
